#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//
// This sample illustrates the usage of CUDA events for both GPU timing and
// overlapping CPU and GPU execution.  Events are inserted into a stream
// of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
// perform computations while GPU is executing (including DMA memcopies
// between the host and device).  CPU can query CUDA events to determine
// whether GPU has completed tasks.
//

// includes, system
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions
#include <hip/hip_vector_types.h>

#define MARKERS  31
#define BLOCK_SIZE 32

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void pointDistance_kernel(float3 *first, float3 *second, float *retVal, int size)
{
    unsigned int tid = threadIdx.x;
    int idx =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    if(idx >= size) return;

    float res =  length(first[idx] - second[idx]);

    retVal[idx] = res;

    __syncthreads();

    unsigned int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
    unsigned int gridSize = BLOCK_SIZE*gridDim.x;

    float *sdata = SharedMemory<float>();

    float mySum = 0.0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < size)
    {
        mySum += retVal[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (i + BLOCK_SIZE < size)
            mySum += retVal[i+BLOCK_SIZE];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;

    __syncthreads();

    if ((BLOCK_SIZE >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((BLOCK_SIZE >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((BLOCK_SIZE >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((BLOCK_SIZE >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((BLOCK_SIZE >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();

    // write result for this block to global mem
    if (tid == 0) retVal[blockIdx.x] = mySum;
}

bool correct_output(int *data, const int n, const int x)
{
    for (int i = 0; i < n; i++)
        if (data[i] != x)
        {
            printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
            return false;
        }

    return true;
}

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void reduce6(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

    __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

extern "C"
int cuda_main(int argc, char *argv[], std::vector<std::vector<std::vector<float3>>> anims)
{
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    std::vector<std::vector<float3>> first = anims[0];
    std::vector<std::vector<float3>> second = anims[1];

    int firstSize = MARKERS * first.size();
    int firstBytes = firstSize * sizeof(float3);

    int secondSize = MARKERS * second.size();
    int secondBytes = secondSize * sizeof(float3);

    float3 *f = &(first[0][0]);
    float3 *s = &(second[0][0]);

    float3 *d_f;
    float3 *d_s;

    // allocate memory
    checkCudaErrors(hipMalloc((void**) &d_f ,firstBytes));
    checkCudaErrors(hipMalloc((void**) &d_s,secondBytes));
    hipMemcpy(d_f, f, firstBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_s, s, secondBytes, hipMemcpyHostToDevice);

    // set kernel launch configuration
    int size = (first.size() < second.size()) ? first.size() : second.size();
    int resultSize = size * MARKERS ;

    dim3 threads(BLOCK_SIZE, MARKERS);
    dim3 blocks(size/BLOCK_SIZE + 1,1 /*first.size()/4*/);

    printf("executing %d blocks result size %d\n", size/BLOCK_SIZE + 1, resultSize);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    float * results;
    checkCudaErrors(hipMallocManaged((void**) &results,resultSize * sizeof(float)));


    // asynchronously issue work to the GPU (all to stream 0)
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    pointDistance_kernel<<<blocks, threads>>>(d_f, d_s,results, resultSize);
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;

    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

    float result = results[0];
    /*for (size_t i = 0; i < resultSize; ++i)
    {
        //printf(" %f ", results[i]);
        result += results[i];
    }*/
    result /=  first.size();

    printf("\n");
    printf("GPU result for size %d: %f\n", size, result);

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(d_f));
    checkCudaErrors(hipFree(d_s));
    checkCudaErrors(hipFree(results));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    return EXIT_SUCCESS;
}

