#include "hip/hip_runtime.h"
// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions
#include <hip/hip_vector_types.h>

#define MARKERS  31


__global__ void pointsMovement(float3 *data, float *firstDifference, float *secondDifference, int size)
{
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    if (idx > size)
    {
        return;
    }

    firstDifference[idx] = length(data[idx + MARKERS] - data[idx]);

    __syncthreads();

    //secondDifference[idx] = length(firstDifference[idx + MARKERS] - firstDifference[idx]);
}
